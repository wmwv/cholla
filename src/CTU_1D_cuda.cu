
#include <hip/hip_runtime.h>
/*! \file CTU_1D_cuda.cu
 *  \brief Definitions of the cuda CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"CTU_1D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_ctu_cuda.h"
#include"plmc_ctu_cuda.h"
#include"ppmp_ctu_cuda.h"
#include"ppmc_ctu_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"cooling_cuda.h"
#include"error_handling.h"
#include"io.h"



__global__ void Update_Conserved_Variables_1D(Real *dev_conserved, Real *dev_F, int n_cells, int n_ghost, 
                                              Real dx, Real dt, Real gamma);

__global__ void Calc_dt_1D(Real *dev_conserved, int n_cells, int n_ghost, Real dx, Real *dti_array, Real gamma);

__global__ void Sync_Energies_1D(Real *dev_conserved, int n_cells, int n_ghost, Real gamma);



Real CTU_Algorithm_1D_CUDA(Real *host_conserved, int nx, int n_ghost, Real dx, Real dt)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid

  // capture the start time
  #ifdef TIME
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  float elapsedTime;
  #endif

  int n_cells = nx;
  int ny = 1;
  int nz = 1;

  int n_fields = 5;
  #ifdef DE
  n_fields = 7;
  #endif

  // set the dimensions of the cuda grid
  int  ngrid = (n_cells + TPB - 1) / TPB;
  dim3 dimGrid(ngrid, 1, 1);
  dim3 dimBlock(TPB, 1, 1);

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(ngrid*sizeof(Real));

  #ifdef TEST
  Real *test1, *test2;
  test1 = (Real *) malloc(5*n_cells*sizeof(Real));
  test2 = (Real *) malloc(5*n_cells*sizeof(Real));
  #endif

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved;
  // initial input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_L, *Q_R, *F;
  // array to hold zero values for H correction (necessary to pass to Roe solver)
  Real *etah;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;


  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_L, n_fields*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_R, n_fields*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F,   (n_fields)*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah, n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );

  // zero all the GPU arrays
  cudaMemset(dev_conserved, 0, n_fields*n_cells*sizeof(Real));
  cudaMemset(Q_L, 0, n_fields*n_cells*sizeof(Real));
  cudaMemset(Q_R, 0, n_fields*n_cells*sizeof(Real));
  cudaMemset(F, 0, n_fields*n_cells*sizeof(Real));
  cudaMemset(etah, 0, n_cells*sizeof(Real));
  cudaMemset(dev_dti_array, 0, ngrid*sizeof(Real));
  CudaCheckError();


  // copy the conserved variable array onto the GPU
  CudaSafeCall( cudaMemcpy(dev_conserved, host_conserved, n_fields*n_cells*sizeof(Real), cudaMemcpyHostToDevice) );
  CudaCheckError();


  // Step 1: Do the reconstruction
  #ifdef PCM
  PCM_Reconstruction_1D<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, n_ghost, gama);
  CudaCheckError();
  #endif
  #ifdef PLMP
  PLMP_CTU<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  CudaCheckError();
  #endif
  #ifdef PLMC
  PLMC_CTU<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  CudaCheckError();
  #endif
  #ifdef PPMP
  PPMP_CTU<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  CudaCheckError();
  #endif
  #ifdef PPMC
  PPMC_CTU<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0);
  CudaCheckError();
  #endif

  
  // Step 2: Calculate the fluxes
  #ifdef EXACT
  Calculate_Exact_Fluxes<<<dimGrid,dimBlock>>>(Q_L, Q_R, F, nx, ny, nz, n_ghost, gama, 0);
  #endif
  #ifdef ROE
  Calculate_Roe_Fluxes<<<dimGrid,dimBlock>>>(Q_L, Q_R, F, nx, ny, nz, n_ghost, gama, etah, 0);
  #endif
  CudaCheckError();


  // Step 3: Update the conserved variable array
  Update_Conserved_Variables_1D<<<dimGrid,dimBlock>>>(dev_conserved, F, n_cells, n_ghost, dx, dt, gama);
  CudaCheckError();
   

  // Sychronize the total and internal energy, if using dual-energy formalism
  #ifdef DE
  Sync_Energies_1D<<<dimGrid,dimBlock>>>(dev_conserved, n_cells, n_ghost, gama);
  #endif


  // Apply cooling
  #ifdef COOLING_GPU
  cooling_kernel<<<dimGrid,dimBlock>>>(dev_conserved, nx, ny, nz, n_ghost, dt, gama);
  #endif

  // Calculate the next timestep
  Calc_dt_1D<<<dimGrid,dimBlock>>>(dev_conserved, n_cells, n_ghost, dx, dev_dti_array, gama);
  CudaCheckError();


  // copy the conserved variable array back to the CPU
  CudaSafeCall( cudaMemcpy(host_conserved, dev_conserved, n_fields*n_cells*sizeof(Real), cudaMemcpyDeviceToHost) );

  // copy the dti array onto the CPU
  CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }


  // free the CPU memory
  free(host_dti_array);

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_L);
  cudaFree(Q_R);
  cudaFree(F);
  cudaFree(etah);

  // return the maximum inverse timestep
  return max_dti;


}


__global__ void Update_Conserved_Variables_1D(Real *dev_conserved, Real *dev_F, int n_cells, int n_ghost, Real dx, Real dt, Real gamma)
{
  int id;
  Real d, d_inv, vx, vy, vz, P;  
  #ifdef DE
  Real vx_imo, vx_ipo;
  #endif

  Real dtodx = dt/dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;

  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    #ifdef DE
    vx_imo = dev_conserved[1*n_cells + id-1]/dev_conserved[id-1];
    vx_ipo = dev_conserved[1*n_cells + id+1]/dev_conserved[id+1];
    #endif
  
    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F[            id-1] - dev_F[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F[  n_cells + id-1] - dev_F[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F[2*n_cells + id-1] - dev_F[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F[3*n_cells + id-1] - dev_F[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F[4*n_cells + id-1] - dev_F[4*n_cells + id]);
    #ifdef DE
    dev_conserved[5*n_cells + id] += dtodx * (dev_F[5*n_cells + id-1] - dev_F[5*n_cells + id])
                                  +  dtodx * P * 0.5 * (vx_imo - vx_ipo);
    #endif
    if (dev_conserved[id] != dev_conserved[id]) printf("%3d Thread crashed in final update.\n", id);
  }


}



__global__ void Sync_Energies_1D(Real *dev_conserved, int n_cells, int n_ghost, Real gamma)
{
  int id;
  Real d, d_inv, vx, vy, vz, P, E;
  Real ge1, ge2, Emax;
  int im1, ip1;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  
  im1 = max(id-1, n_ghost);
  ip1 = min(id+1, n_cells-n_ghost-1);

  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    // separately tracked internal energy 
    ge1 = dev_conserved[5*n_cells + id];
    // internal energy calculated from total energy
    ge2 = dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz);
    // if the ratio of conservatively calculated internal energy to total energy
    // is greater than 1/1000, use the conservatively calculated internal energy
    // to do the internal energy update
    if (ge2/E > 0.001) {
      dev_conserved[5*n_cells + id] = ge2;
      ge1 = ge2;
    }     
    // find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + im1], E);
    Emax = fmax(dev_conserved[4*n_cells + ip1], Emax);
    // if the ratio of conservatively calculated internal energy to max nearby total energy
    // is greater than 1/10, continue to use the conservatively calculated internal energy 
    if (ge2/Emax > 0.1) {
      dev_conserved[5*n_cells + id] = ge2;
    }
    // sync the total energy with the internal energy 
    else {
      dev_conserved[4*n_cells + id] += ge1 - ge2;
    }
     
    // recalculate the pressure 
    P = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);    
    if (P < 0.0) printf("%d Negative pressure after internal energy sync. %f %f \n", id, ge1, ge2);    
  }

}


__global__ void Calc_dt_1D(Real *dev_conserved, int n_cells, int n_ghost, Real dx, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, P, cs;
  int id, tid;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  // and a thread id within the block
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    // start timestep calculation here
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    P  = fmax(P, (Real) TINY_NUMBER);
    // find the max wavespeed in that cell, use it to calculate the inverse timestep
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = (fabs(vx)+cs)/dx;
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockIdx.x] = max_dti[0];


}





#endif //CUDA